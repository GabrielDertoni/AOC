#include <iostream>
#include <hip/hip_runtime.h>

class Matrix {
public:

    __device__ __host__ int& get(int i, int j) {
        return data[i * width + j];
    }

private:
    int *data;
    int width;
    int height;
};

__global__ void compute_permutations(Matrix graph, int *best) {
}

int permutation_flow(Matrix graph, int *flows, int *path, int n) {
    int flow = 0;
    int turns = 0;
    for (int i = 0; i < n && turns < 30; i++) {
        if (i == 0)
            turns += graph.get(0, path[i]) + 1;
        else
            turns += graph.get(path[i - 1], path[i]) + 1;

        flow += flows[path[i]] * (30 - turns);
    }
    return flow;
}

int main() {
    return 0;
}
